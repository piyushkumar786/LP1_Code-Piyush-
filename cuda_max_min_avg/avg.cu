#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 100

__global__ void max_kernel(int *a, int *n) {
    int tid = threadIdx.x;
    int start = *n * tid;
    int end = *n * (tid + 1);
    
    for (int i = start; i < end; i++) {    
            a[start] += a[i];
    }
}

int main() {
    
    srand(time(NULL));
	int *arr, *d_arr, *dev_n;
	int n = 5;   
	arr = new int[N];
	
   	for (int i = 0; i < N; i++) {
        arr[i] = rand() % 1000;
	}
	
	hipMalloc(&d_arr, N * sizeof(int));
	hipMalloc(&dev_n, sizeof(int));
	
	hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
	
	int len = N;
    
    while (len > n) {
    
        max_kernel<<<1, len / n>>>(d_arr, dev_n);
        
        hipDeviceSynchronize();
        
        hipMemcpy(arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);
        
        printf("Pass %d: ", pass++);
        for (int i = 0; i < (len/n); i++) {
            arr[i] = arr[i * n];
            printf("%d ", arr[i]);
        }
        printf("\n");
        
    	hipMemcpy(d_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    	
        len /= n;
    }
    
    for (int i = 0; i < len; i++) {        
            arr[0] += arr[i];
    }
    
    printf("average : %d\n", arr[0] / 100);
    
    return 0;
}
