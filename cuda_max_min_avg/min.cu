#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>
#define SIZE 100
__global__ void  min(int *a,int *c)
{
	int i =threadIdx.a;
	if(a[i]<*c)
		c=a[i];

}

int main()
{
	
	int i ;
	srand(time(NULL));
	int *dev_a,*dev_c;

	int a[SIZE],int c;
	hipMalloc((void**)&dev_a, SIZE*sizeof(int));
	hipMalloc((void**)&dev_c, SIZE*sizeof(int));
	for(i=0;i<SIZE;i++)
		a[i]=i;

**dev_c =100;
hipMemcpy(dev_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
min<<<(1,SIZE>>>(dev_a,dev_c);
hipMemcpy(&c,dev_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);
printf("min=%d",c);
hipFree(dev_a);
hipFree(dev_c);

}
