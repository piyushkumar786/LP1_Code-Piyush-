#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>
#define SIZE 1000
__global__ void  min(int *a,int *c)
{
	int i =threadIdx.a;
	if(a[i]>*c)
		c=a[i];

}

int main()
{
	
	int i ;
	srand(time(NULL));
	int *dev_a,*dev_c;

	int a[SIZE],int c;
	hipMalloc((void**)&dev_a, SIZE*sizeof(int));
	hipMalloc((void**)&dev_c, SIZE*sizeof(int));
	for(i=0;i<SIZE;i++)
		a[i]=rand()%1000+1;

**dev_c =0;
hipMemcpy(dev_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
min<<<(1,SIZE>>>(dev_a,dev_c);
hipMemcpy(&c,dev_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);
printf("max=%d",c);
hipFree(dev_a);
hipFree(dev_c);

}
